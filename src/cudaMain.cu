#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////
// MIT License
//
//Copyright (c) 2019 - 2021 Iowa State University
//
//Permission is hereby granted, free of charge, to any person obtaining a copy
//of this software and associated documentation files (the "Software"), to deal
//in the Software without restriction, including without limitation the rights
//to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
//copies of the Software, and to permit persons to whom the Software is
//furnished to do so, subject to the following conditions:
//
//The above copyright notice and this permission notice shall be included in all
//copies or substantial portions of the Software.
//
//THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
//AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
//OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
//SOFTWARE.
//////////////////////////////////////////////////////////////////////////////////


#include <cudaMain.h>
#include <Input/Input.h>
#include <bits/ios_base.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <Output/writeVTI.h>
#include <uniaxial.h>
#include <hipblas.h>
#include <chrono>
#include <ctime>
#include <chrono>
#include <npp.h>
#include <Output/outputUtils.h>
//#include <RotationMatrix.h>
#define START_TIMER(X) if(ompThreadID == 0){timerArrayStart[X] = std::chrono::high_resolution_clock::now();}
#define END_TIMER(X) if(ompThreadID == 0){timerArrayEnd[X] = std::chrono::high_resolution_clock::now(); \
                     timings[X] +=  (static_cast<std::chrono::duration<Real>>(timerArrayEnd[X] - timerArrayStart[X])).count();}


int warmup() {
  double *d_warmup, *warmup;
  warmup = new double[1000];
  CUDA_CHECK_RETURN(hipMalloc((void **) &d_warmup, sizeof(double) * 1000));
  gpuErrchk(hipPeekAtLastError());
  CUDA_CHECK_RETURN(hipMemcpy(d_warmup, warmup, sizeof(double) * 1000, hipMemcpyHostToDevice));
  gpuErrchk(hipPeekAtLastError());
  hipFree(d_warmup);
  delete[] warmup;
  return EXIT_SUCCESS;
}

__host__ int performFFTShift(Complex *polarization, const UINT &blockSize, const uint3 &vx, const hipStream_t stream) {
  FFTIgor<<<blockSize, NUM_THREADS,0,stream>>>(polarization, vx);
  return EXIT_SUCCESS;
}

__host__  int performScatter3DComputation(const Complex *d_polarizationX, const Complex *d_polarizationY,
                                          const Complex *d_polarizationZ,
                                          Real *d_scatter3D,
                                          const Real & kMagnitude,
                                          const BigUINT &voxelSize,
                                          const uint3 &vx,
                                          const Real &physSize,
                                          const bool &enable2D,
                                          const UINT &blockSize,
                                          const Real3 & kVector) {

  computeScatter3D <<< blockSize, NUM_THREADS >>>(d_polarizationX, d_polarizationY, d_polarizationZ,
                                                  d_scatter3D,  kMagnitude , voxelSize, vx,
                                                  physSize,
                                                  enable2D, kVector);
  hipDeviceSynchronize();
  gpuErrchk(hipPeekAtLastError());
  return EXIT_SUCCESS;
}

__host__ int peformEwaldProjectionGPU(Real *d_projection,
                                      const Real *d_scatter,
                                      const Real & kMagnitude,
                                      const uint3 &vx,
                                      const Real &physSize,
                                      const Interpolation::EwaldsInterpolation &interpolation,
                                      const bool &enable2D,
                                      const UINT &blockSize,
                                      const Real3 & kVector) {
  computeEwaldProjectionGPU <<< blockSize, NUM_THREADS >>>(d_projection, d_scatter, vx,
                                                           kMagnitude, physSize,
                                                           interpolation,
                                                           enable2D,kVector);
  hipDeviceSynchronize();
  gpuErrchk(hipPeekAtLastError());

  return EXIT_SUCCESS;


}

__host__ int peformEwaldProjectionGPU(Real *d_projection,
                                      const Complex *d_polarizationX, const Complex *d_polarizationY,
                                      const Complex *d_polarizationZ,
                                      const Real &kMagnitude,
                                      const uint3 &vx,
                                      const Real &physSize,
                                      const Interpolation::EwaldsInterpolation &interpolation,
                                      const bool &enable2D,
                                      const UINT &blockSize,
                                      const Real3 & kVector) {
  computeEwaldProjectionGPU <<< blockSize, NUM_THREADS >>>(d_projection, d_polarizationX, d_polarizationY,
                                                           d_polarizationZ, vx,
                                                           kMagnitude, physSize,
                                                           interpolation,
                                                           enable2D,kVector);
  hipDeviceSynchronize();
  gpuErrchk(hipPeekAtLastError());
  return EXIT_SUCCESS;

}

template<ReferenceFrame referenceFrame>
__global__ void computePolarization(const Material * d_materialConstants,
                                    const Voxel *voxelInput,
                                    const uint3 voxel,
                                    Complex *polarizationX,
                                    Complex *polarizationY,
                                    Complex *polarizationZ,
                                    FFT::FFTWindowing windowing,
                                    const bool enable2D,
                                    const MorphologyType morphologyType,
                                    const Matrix rotationMatrix,
                                    const BigUINT numVoxels, const int DEVICE_NUM_MATERIAL
) {
  BigUINT threadID = threadIdx.x + blockIdx.x * blockDim.x;
  if(threadID > numVoxels){
    return;
  }
#ifndef BIAXIAL
  if (morphologyType == MorphologyType::VECTOR_MORPHOLOGY) {
    computePolarizationVectorMorphologyOptimized<referenceFrame>(d_materialConstants, voxelInput, threadID, polarizationX,
                                                 polarizationY,
                                                 polarizationZ,numVoxels,rotationMatrix,DEVICE_NUM_MATERIAL);
  } else {
    computePolarizationEulerAngles<referenceFrame>(d_materialConstants, voxelInput, threadID, polarizationX, polarizationY,
                                   polarizationZ,numVoxels,rotationMatrix,DEVICE_NUM_MATERIAL);
  }
#else
  printf("Kernel not spported\n");
#endif


  if (windowing == FFT::FFTWindowing::HANNING) {
    UINT Z = static_cast<UINT>(threadID / (voxel.y * voxel.x * 1.0));
    UINT Y = static_cast<UINT>((threadID - Z * voxel.y * voxel.x) / (voxel.x * 1.0));
    UINT X = static_cast<UINT>(threadID - Y * voxel.x - Z * voxel.y * voxel.x);
    Real3 hanningWeight;
    hanningWeight.x = static_cast<Real> (0.5 * (1 - cos(2 * M_PI * X / (voxel.x))));
    hanningWeight.y = static_cast<Real> (0.5 * (1 - cos(2 * M_PI * Y / (voxel.y))));
    hanningWeight.z = static_cast<Real>(1.0);
    if (not(enable2D)) {
      hanningWeight.z = static_cast<Real>(0.5 * (1 - cos(2 * M_PI * Z / (voxel.z))));
    }
    Real totalHanningWeight = hanningWeight.x * hanningWeight.y * hanningWeight.z;
    polarizationX[threadID].x *= totalHanningWeight;
    polarizationX[threadID].y *= totalHanningWeight;
    polarizationY[threadID].x *= totalHanningWeight;
    polarizationY[threadID].y *= totalHanningWeight;
    polarizationZ[threadID].x *= totalHanningWeight;
    polarizationZ[threadID].y *= totalHanningWeight;

  }

}

__host__ int computePolarization(const Material  * d_materialConstants,
                                 const Voxel *d_voxelInput,
                                 const uint3 &vx,
                                 Complex *d_polarizationX,
                                 Complex *d_polarizationY,
                                 Complex *d_polarizationZ,
                                 const FFT::FFTWindowing & windowing,
                                 const bool &enable2D,
                                 const MorphologyType &morphologyType,
                                 const UINT &blockSize,
                                 const ReferenceFrame & referenceFrame,
                                 const Matrix & rotationMatrix,
                                 const BigUINT & numVoxels,const int NUM_MATERIAL
) {
  if(referenceFrame == ReferenceFrame::MATERIAL) {
    computePolarization<ReferenceFrame::MATERIAL><<< blockSize, NUM_THREADS >>>(d_materialConstants, d_voxelInput,
                                                                                 vx, d_polarizationX,
                                                                                d_polarizationY, d_polarizationZ,
                                                                                windowing,
                                                                                enable2D,
                                                                                morphologyType,rotationMatrix, numVoxels,NUM_MATERIAL);
  }
  else  {
    computePolarization<ReferenceFrame::LAB><<< blockSize, NUM_THREADS >>>(d_materialConstants, d_voxelInput,
                                                                                 vx, d_polarizationX,
                                                                                d_polarizationY, d_polarizationZ,
                                                                                windowing,
                                                                                enable2D,
                                                                                morphologyType,rotationMatrix,numVoxels,NUM_MATERIAL);
  }
  hipDeviceSynchronize();
  gpuErrchk(hipPeekAtLastError());
  return EXIT_SUCCESS;
}

__host__ int computeNt(const Material * d_materialConstants,
                       const Voxel *d_voxelInput,
                       Complex * d_Nt,
                       const MorphologyType &morphologyType,
                       const UINT &blockSize,
                       const BigUINT & numVoxels,
                       const BigUINT & offset,
                       const BigUINT & endID,
                       const UINT & materialID,
                       const UINT & numStreams,
                       hipStream_t stream, int NUM_MATERIAL

) {

  if(morphologyType == MorphologyType::VECTOR_MORPHOLOGY) {
    computeNtVectorMorphology<<<std::ceil(blockSize*1.0/numStreams), NUM_THREADS,0,stream>>>(d_materialConstants, d_voxelInput, d_Nt, offset,endID,materialID,numVoxels,NUM_MATERIAL);
  } else{
    computeNtEulerAngles<<<std::ceil(blockSize*1.0/numStreams), NUM_THREADS,0,stream>>>(d_materialConstants, d_voxelInput, d_Nt, offset,endID,materialID,numVoxels,NUM_MATERIAL);
  }
  return (EXIT_SUCCESS);
}

__host__ int computePolarization(const Complex * __restrict__ d_Nt, Complex *d_pX,
                                 Complex *d_pY, Complex *d_pZ,
                                 const UINT &blockSize,
                                 const ReferenceFrame &referenceFrame,
                                 const Matrix &rotationMatrix,
                                 const BigUINT &numVoxels

) {
  if (referenceFrame == ReferenceFrame::MATERIAL) {
      computePolarizationVectorMorphologyLowMemory<ReferenceFrame::MATERIAL><<<blockSize, NUM_THREADS >>>( (Real4 *)d_Nt, d_pX,
                                                                                                          d_pY, d_pZ,rotationMatrix,numVoxels);
    } else{
      computePolarizationVectorMorphologyLowMemory<ReferenceFrame::LAB><<<blockSize, NUM_THREADS >>>((Real4 *)d_Nt, d_pX,
                                                                                                     d_pY, d_pZ,rotationMatrix,numVoxels);
    }

    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    return EXIT_SUCCESS;
  }

int cudaMain(const UINT *voxel,
             const InputData &idata,
             const std::vector<Material>  &materialInput,
             Real *projectionGPUAveraged,
             RotationMatrix & rotationMatrix,
             const Voxel *voxelInput) {


  if ((static_cast<uint64_t>(voxel[0]) * voxel[1] * voxel[2]) > std::numeric_limits<BigUINT>::max()) {
    std::cout << "[Compile error] Exiting. Compile by Enabling 64 Bit indices\n";
    exit(EXIT_FAILURE);
  }

  const BigUINT numVoxels = voxel[0] * voxel[1] * voxel[2]; /// Voxel size
  const UINT numVoxel2D = voxel[0] * voxel[1];
  const uint3 vx{voxel[0], voxel[1], voxel[2]};
  const UINT
    numAnglesRotation = static_cast<UINT>(std::round((idata.endAngle - idata.startAngle) / idata.incrementAngle + 1));
  const UINT &numEnergyLevel = idata.energies.size();

  const int & NUM_MATERIAL = idata.NUM_MATERIAL;
  int num_gpu;
  hipGetDeviceCount(&num_gpu);
  std::cout << "Number of CUDA devices:" << num_gpu << "\n";

  if (num_gpu < 1) {
    std::cout << "[GPU error] No GPU found. Exiting" << "\n";
    return (EXIT_FAILURE);
  }

#ifdef PROFILING
  enum TIMERS:UINT{
    MALLOC = 0,
    MEMCOPY_CPU_GPU = 1,
    POLARIZATION = 2,
    FFT = 3,
    SCATTER3D = 4,
    IMAGE_ROTATION=5,
    MEMCOPY_GPU_CPU = 6,
    ENERGY=7,
    MAX = 8
  };
  static const char *timersName[]{"Malloc on CPU + GPU",
                                  "Memcopy CPU -> GPU",
                                  "Polarization",
                                  "FFT",
                                  "Scatter3D + Ewalds",
                                  "Rotation",
                                  "Memcopy GPU -> CPU",
                                  "Total time "};
  static_assert(sizeof(timersName) / sizeof(char*) == TIMERS::MAX,
                "sizes dont match");
  std::array<std::chrono::high_resolution_clock::time_point,TIMERS::MAX> timerArrayStart;
  std::array<std::chrono::high_resolution_clock::time_point,TIMERS::MAX> timerArrayEnd;
  std::array<Real,TIMERS::MAX> timings{};
  timings.fill(0.0);

#endif

#ifdef DUMP_FILES
  createDirectory("Polarize");
  createDirectory("FFT");
  createDirectory("Scatter");
  createDirectory("Ewald");

  /** Writing VTI files as a cross check **/

  const char * varnameVector[4] = {"material1_s","material2_s","material3_s","material4_s"};

  const char * varnameScalar[4] = {"phi0","phi1", "phi2", "phi3"};

  VTI::writeVoxelDataVector(voxelInput, voxel, "S1", varnameVector,NUM_MATERIAL);
  VTI::writeVoxelDataScalar(voxelInput, voxel, "Phi", varnameScalar,NUM_MATERIAL);
#endif
  omp_set_num_threads(num_gpu);
#pragma omp parallel
  {


    hipSetDevice(omp_get_thread_num());
    hipDeviceProp_t dprop;
    hipGetDeviceProperties(&dprop, omp_get_thread_num());

#ifdef PROFILING
    if(warmup() == EXIT_SUCCESS){
      std::cout << "Warmup completed on GPU " << dprop.name << "\n";
    }
    else{
      std::cout << "Warmup failed on GPU " << dprop.name << "\n";
#pragma omp cancel parallel
      exit (EXIT_FAILURE);
    }
#endif
    static constexpr int NUM_STREAMS=3;
    hipStream_t streams[NUM_STREAMS];
    hipfftResult result[NUM_STREAMS];
    hipfftHandle plan[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
      gpuErrchk(hipStreamCreate(&streams[i]));
    }

    for(int i = 0; i < NUM_STREAMS; i++){
      hipfftPlan3d(&plan[i], voxel[2], voxel[1], voxel[0], fftType);
      hipfftSetStream(plan[i],streams[i]);
    }

    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hipblasCreate(&handle);

    NppiSize sizeImage;
    sizeImage.height = voxel[0];
    sizeImage.width = voxel[1];

    NppiRect rect;
    rect.height = voxel[0];
    rect.width = voxel[1];
    rect.x = 0;
    rect.y = 0;


    const UINT ompThreadID = omp_get_thread_num();
    const UINT numEnergyPerGPU = static_cast<UINT>(std::ceil(numEnergyLevel * 1.0 / num_gpu));
    const UINT numStart = (numEnergyPerGPU * ompThreadID);
    UINT numEnd = (numEnergyPerGPU * (ompThreadID + 1));
    numEnd = std::min(numEnd, numEnergyLevel);

    const Real &energyStart = numStart < numEnergyLevel ? idata.energies[numStart] : 0;
    const Real &energyEnd = idata.energies[numEnd - 1];

    if (numStart >= numEnergyLevel) {
      std::cout << "[INFO] [GPU = " << dprop.name << "] -> No computation. Idle\n";
    } else {
      std::cout << "[INFO] [GPU = " << dprop.name << "] : " << energyStart << "eV -> " << energyEnd << "eV\n";
    }


#ifdef PROFILING
    {
      START_TIMER(TIMERS::MALLOC);
    }
#endif
#ifdef DUMP_FILES
    Complex *polarizationZ = new Complex[numVoxels];
    Complex *polarizationX = new Complex[numVoxels];
    Complex *polarizationY = new Complex[numVoxels];
#endif
#if defined(EOC) or defined(DUMP_FILES)
    Real *scatter3D = new Real[numVoxels];
#endif

#ifdef EOC
    Real *projectionCPU = new Real[BATCH * voxel[0] * voxel[1]];
#else

#endif

    Voxel *d_voxelInput;
    mallocGPU(d_voxelInput, numVoxels*NUM_MATERIAL);

    Complex *d_polarizationZ, *d_polarizationX, *d_polarizationY;
    Real *d_scatter3D;
    UINT *d_mask;
    Material * d_materialConstants;

    mallocGPU(d_polarizationX, numVoxels);
    mallocGPU(d_polarizationY, numVoxels);
    mallocGPU(d_polarizationZ, numVoxels);
    mallocGPU(d_materialConstants, NUM_MATERIAL);

    if (idata.scatterApproach == ScatterApproach::FULL) {
      mallocGPU(d_scatter3D, numVoxels);
    }
#ifndef EOC
    Real *d_projection, *d_rotProjection, *d_projectionAverage;
    mallocGPU(d_projection, numVoxel2D);
    mallocGPU(d_rotProjection, numVoxel2D);
    if (idata.rotMask) {
      mallocGPU(d_mask, numVoxel2D);
    }
    mallocGPU(d_projectionAverage, numVoxel2D);
#endif


#ifdef PROFILING
    {
      END_TIMER(TIMERS::MALLOC)
      START_TIMER(TIMERS::MEMCOPY_CPU_GPU)
    }
#endif

    hostDeviceExchange(d_voxelInput, voxelInput, numVoxels*NUM_MATERIAL, hipMemcpyHostToDevice);
#ifdef PROFILING
    {
      END_TIMER(TIMERS::MEMCOPY_CPU_GPU)
    }
#endif
    // TODO: Make this async and overlap with computation
    rotationMatrix.initComputation();
    const auto & baseConfigurations = rotationMatrix.getBaseConfigurations();


    const auto & kVectors = idata.kVectors;


    UINT BlockSize  = static_cast<UINT>(ceil(numVoxels * 1.0 / NUM_THREADS));
    UINT BlockSize2 = static_cast<UINT>(ceil(numVoxel2D * 1.0 / NUM_THREADS));

    for (UINT j = numStart; j < numEnd; j++) {

      hostDeviceExchange(d_materialConstants, &materialInput[j * NUM_MATERIAL], NUM_MATERIAL, hipMemcpyHostToDevice);
      const Real &energy = (idata.energies[j]);
      std::cout << " [STAT] Energy = " << energy << " starting " << "\n";
      for (UINT kstart = 0; kstart < kVectors.size(); kstart++) {
        const auto & baseConfig = baseConfigurations[kstart];
        const Real baseRotAngle = baseConfig.baseRotAngle;
        const Matrix & rotationMatrixK = baseConfig.matrix;
        const Real3 &kVec = idata.kVectors[kstart];
        cudaZeroEntries(d_projectionAverage, numVoxel2D);
        if (idata.rotMask) {
          cudaZeroEntries(d_mask, numVoxel2D);
        }

#ifdef  PROFILING
        START_TIMER(TIMERS::ENERGY)
#endif



        const Real wavelength = static_cast<Real>(1239.84197 / energy);
        const Real kMagnitude = static_cast<Real>(2 * M_PI / wavelength);;
        Real Eangle;
        Matrix ERotationMatrix;
        for (UINT i = 0; i < numAnglesRotation; i++) {
          Eangle = static_cast<Real>((baseRotAngle + idata.startAngle + i * idata.incrementAngle) * M_PI / 180.0);
          computeRotationMatrix(kVec, rotationMatrixK, ERotationMatrix, Eangle);
#ifdef PROFILING
          {
            START_TIMER(TIMERS::POLARIZATION)
          }
#endif
          computePolarization(d_materialConstants, d_voxelInput, vx, d_polarizationX, d_polarizationY,
                              d_polarizationZ, static_cast<FFT::FFTWindowing >(idata.windowingType),
                              idata.if2DComputation(), static_cast<MorphologyType>(idata.morphologyType), BlockSize,
                              static_cast<ReferenceFrame>(idata.referenceFrame), ERotationMatrix, numVoxels,idata.NUM_MATERIAL);

#ifdef DUMP_FILES

          CUDA_CHECK_RETURN(hipMemcpy(polarizationX,
                                       d_polarizationX,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          CUDA_CHECK_RETURN(hipMemcpy(polarizationZ,
                                       d_polarizationZ,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          CUDA_CHECK_RETURN(hipMemcpy(polarizationY,
                                       d_polarizationY,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          {
            FILE *pX = fopen("polarizeX.dmp", "wb");
            fwrite(polarizationX, sizeof(Complex), numVoxels, pX);
            fclose(pX);
            FILE *pY = fopen("polarizeY.dmp", "wb");
            fwrite(polarizationY, sizeof(Complex), numVoxels, pY);
            fclose(pY);
            FILE *pZ = fopen("polarizeZ.dmp", "wb");
            fwrite(polarizationZ, sizeof(Complex), numVoxels, pZ);
            fclose(pZ);
            std::string dirname = "Polarize/";
            std::string fname = dirname + "polarizationX" + std::to_string(i);
            VTI::writeDataScalar(polarizationX, voxel, fname.c_str(), "polarizeX");
            fname = dirname + "polarizationY" + std::to_string(i);
            VTI::writeDataScalar(polarizationY, voxel, fname.c_str(), "polarizeY");
            fname = dirname + "polarizationZ" + std::to_string(i);
            VTI::writeDataScalar(polarizationZ, voxel, fname.c_str(), "polarizeZ");
          }
#endif

#ifdef PROFILING
          {
            END_TIMER(TIMERS::POLARIZATION)
            START_TIMER(TIMERS::FFT)
          }
#endif
          /** FFT Computation **/
          result[0] = performFFT(d_polarizationX, plan[0]);
          result[1] = performFFT(d_polarizationY, plan[1]);
          result[2] = performFFT(d_polarizationZ, plan[2]);


#ifdef DUMP_FILES
          CUDA_CHECK_RETURN(hipMemcpy(polarizationX,
                                       d_polarizationX,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          CUDA_CHECK_RETURN(hipMemcpy(polarizationY,
                                       d_polarizationY,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          CUDA_CHECK_RETURN(hipMemcpy(polarizationZ,
                                       d_polarizationZ,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          {
            FILE *pX = fopen("fftpolarizeXbshift.dmp", "wb");
            fwrite(polarizationX, sizeof(Complex), numVoxels, pX);
            fclose(pX);
            FILE *pY = fopen("fftpolarizeYbshift.dmp", "wb");
            fwrite(polarizationY, sizeof(Complex), numVoxels, pY);
            fclose(pY);
            FILE *pZ = fopen("fftpolarizeZbshift.dmp", "wb");
            fwrite(polarizationZ, sizeof(Complex), numVoxels, pZ);
            fclose(pZ);
            std::string dirname = "FFT/";
            std::string fname = dirname + "polarizationXfftbshift" + std::to_string(i);
            VTI::writeDataScalar(polarizationX, voxel, fname.c_str(), "polarizeXfft");
            fname = dirname + "polarizationYfftbshift" + std::to_string(i);
            VTI::writeDataScalar(polarizationY, voxel, fname.c_str(), "polarizeYfft");
            fname = dirname + "polarizationZfftbshift" + std::to_string(i);
            VTI::writeDataScalar(polarizationZ, voxel, fname.c_str(), "polarizeZfft");
          }
#endif
          performFFTShift(d_polarizationX, BlockSize, vx,streams[0]);
          performFFTShift(d_polarizationY, BlockSize, vx,streams[1]);
          performFFTShift(d_polarizationZ, BlockSize, vx,streams[2]);
          hipDeviceSynchronize();
          gpuErrchk(hipPeekAtLastError());
          if ((result[0] != HIPFFT_SUCCESS) or (result[1] != HIPFFT_SUCCESS) or (result[2] != HIPFFT_SUCCESS)) {
            std::cout << "CUFFT failed with result " << result[0] << " " << result[1] << " " << result[2] << "\n";
#pragma omp cancel parallel
            exit(EXIT_FAILURE);
          }
#ifdef DUMP_FILES
          CUDA_CHECK_RETURN(hipMemcpy(polarizationX,
                                       d_polarizationX,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          CUDA_CHECK_RETURN(hipMemcpy(polarizationY,
                                       d_polarizationY,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          CUDA_CHECK_RETURN(hipMemcpy(polarizationZ,
                                       d_polarizationZ,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          {
            FILE *pX = fopen("fftpolarizeX.dmp", "wb");
            fwrite(polarizationX, sizeof(Complex), numVoxels, pX);
            fclose(pX);
            FILE *pY = fopen("fftpolarizeY.dmp", "wb");
            fwrite(polarizationY, sizeof(Complex), numVoxels, pY);
            fclose(pY);
            FILE *pZ = fopen("fftpolarizeZ.dmp", "wb");
            fwrite(polarizationZ, sizeof(Complex), numVoxels, pZ);
            fclose(pZ);
            std::string dirname = "FFT/";
            std::string fname = dirname + "polarizationXfft" + std::to_string(i);
            VTI::writeDataScalar(polarizationX, voxel, fname.c_str(), "polarizeXfft");
            fname = dirname + "polarizationYfft" + std::to_string(i);
            VTI::writeDataScalar(polarizationY, voxel, fname.c_str(), "polarizeYfft");
            fname = dirname + "polarizationZfft" + std::to_string(i);
            VTI::writeDataScalar(polarizationZ, voxel, fname.c_str(), "polarizeZfft");
          }
#endif

#ifdef PROFILING
          {
              END_TIMER(TIMERS::FFT)
              START_TIMER(TIMERS::SCATTER3D)
          }
#endif
          cudaZeroEntries(d_rotProjection, numVoxel2D);
          cudaZeroEntries(d_projection, numVoxel2D);

          if (idata.scatterApproach == ScatterApproach::FULL) {

            performScatter3DComputation(d_polarizationX, d_polarizationY, d_polarizationZ, d_scatter3D, kMagnitude,
                                        numVoxels, vx, idata.physSize, idata.if2DComputation(), BlockSize, kVec);

#ifdef DUMP_FILES
            CUDA_CHECK_RETURN(hipMemcpy(scatter3D, d_scatter3D, sizeof(Real) * numVoxels, hipMemcpyDeviceToHost));
            gpuErrchk(hipPeekAtLastError())
            {
              FILE *scatter = fopen("scatter_3D.dmp", "wb");
              fwrite(scatter3D, sizeof(Real), numVoxels, scatter);
              fclose(scatter);
              std::string dirname = "Scatter/";
              std::string fname = dirname + "scatter" + std::to_string(i);
              VTI::writeDataScalar(scatter3D, voxel, fname.c_str(), "scatter3D");
            }

#endif


#ifdef EOC
            CUDA_CHECK_RETURN(hipMemcpy(scatter3D, d_scatter3D, sizeof(Real) * numVoxels, hipMemcpyDeviceToHost));
            gpuErrchk(hipPeekAtLastError());

#ifdef PROFILING
            {

            }
#endif
            computeEwaldProjectionCPU(projectionCPU, scatter3D, vx, eleField.k.x);
#else
            peformEwaldProjectionGPU(d_projection, d_scatter3D, kMagnitude, vx, idata.physSize,
                                     static_cast<Interpolation::EwaldsInterpolation>(idata.ewaldsInterpolation),
                                     idata.if2DComputation(), BlockSize2, kVec);
#ifdef DUMP_FILES
            hostDeviceExchange(projectionGPUAveraged, d_projection, voxel[0] * voxel[1], hipMemcpyDeviceToHost);
            std::string dirname = "Ewald/";
            std::string fname = dirname + "ewlad" + std::to_string(i);
            VTI::writeDataScalar2DFP(projectionGPUAveraged, voxel, fname.c_str(), "ewald");
            FILE *projection = fopen("projection_scatterFull.dmp", "wb");
            fwrite(projectionGPUAveraged, sizeof(Real), numVoxels, projection);
            fclose(projection);
#endif
          } else {
            peformEwaldProjectionGPU(d_projection, d_polarizationX, d_polarizationY, d_polarizationZ,kMagnitude,
                                      vx,idata.physSize,
                                     static_cast<Interpolation::EwaldsInterpolation>(idata.ewaldsInterpolation),
                                     idata.if2DComputation(), BlockSize2, kVec);
#ifdef DUMP_FILES

            hostDeviceExchange(projectionGPUAveraged, d_projection, voxel[0] * voxel[1], hipMemcpyDeviceToHost);
            std::string dirname = "Ewald/";
            std::string fname = dirname + "ewlad" + std::to_string(i);
            VTI::writeDataScalar2DFP(projectionGPUAveraged, voxel, fname.c_str(), "ewald");
            FILE *projection = fopen("projection_scatterPartial.dmp", "wb");
            fwrite(projectionGPUAveraged, sizeof(Real), numVoxels, projection);
            fclose(projection);
#endif
          }


          Real _factor;
          _factor = NAN;

          stat = cublasScale(handle, numVoxel2D, &_factor, d_rotProjection, 1);


          if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS during scaling failed  with status " << stat << "\n";
            exit(EXIT_FAILURE);
          }


#ifdef PROFILING
          {
            END_TIMER(TIMERS::SCATTER3D)
            START_TIMER(TIMERS::IMAGE_ROTATION)
          }
#endif
          const double alpha = cos(-Eangle);
          const double beta = sin(-Eangle);

          /**https://docs.opencv.org/2.4/modules/imgproc/doc/geometric_transformations.html?highlight=warpaffine**/
          const double coeffs[2][3]{
            alpha, beta, static_cast<Real>(((1 - alpha) * voxel[0] / 2 - beta * voxel[1] / 2.)),
            -beta, alpha, static_cast<Real>(beta * voxel[0] / 2. + (1 - alpha) * voxel[1] / 2.)
          };


          NppStatus status = warpAffine(d_projection,
                                        sizeImage,
                                        voxel[1] * sizeof(Real),
                                        rect,
                                        d_rotProjection,
                                        voxel[1] * sizeof(Real),
                                        rect,
                                        coeffs,
                                        NPPI_INTER_LINEAR);

          if (status < 0) {
            std::cout << "Image rotation failed with error = " << status << "\n";
            exit(-1);
          }
          if (status != NPP_SUCCESS) {
            std::cout << YLW << "[WARNING] Image rotation warning = " << status << NRM << "\n";
          }

          if (idata.rotMask) {
            computeRotationMask<<< BlockSize2, NUM_THREADS >>>(d_rotProjection, d_mask, vx);
            hipDeviceSynchronize();
          }

          const Real factor = static_cast<Real>(1.0);
          stat = cublasAXPY(handle, numVoxel2D, &factor, d_rotProjection, 1, d_projectionAverage, 1);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS during sum failed  with status " << stat << "\n";
            exit(EXIT_FAILURE);
          }

#ifdef PROFILING
          {
            END_TIMER(TIMERS::IMAGE_ROTATION)
          }
#endif
#endif
        }

        if (idata.rotMask) {
          averageRotation<<<BlockSize2, NUM_THREADS>>>(d_projectionAverage, d_mask, vx);
          hipDeviceSynchronize();
          gpuErrchk(hipPeekAtLastError());
        } else {
          /// The averaging out for all angles
          const Real alphaFac = static_cast<Real>(1.0 / numAnglesRotation);
          stat = cublasScale(handle, voxel[0] * voxel[1], &alphaFac, d_projectionAverage, 1);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS during averaging failed  with status " << stat << "\n";
            exit(EXIT_FAILURE);
          }
        }

#ifdef PROFILING
        {
          START_TIMER(TIMERS::IMAGE_ROTATION)
        }
#endif
        //// Rotate Image
        hostDeviceExchange(d_projection, d_projectionAverage, numVoxel2D, hipMemcpyDeviceToDevice);
        const double srcPoints[3][2]{{voxel[0] / 2.,  voxel[1] / 2.},
                                     {voxel[0] * 0.5, voxel[1] * 1.0},
                                     {voxel[0] * 1.0, voxel[1] * 0.5}};
        Real3 _dstPts[3], _srcPts;
        double center[2]{voxel[0] / 2., voxel[1] / 2.};
        for (int i = 0; i < 3; i++) {
          _srcPts.x = srcPoints[i][0] - center[0];
          _srcPts.y = srcPoints[i][1] - center[1];
          _srcPts.z = 0;
          const Matrix & detectorMatrix = rotationMatrix.getDetectorRotationMatrix();
          Matrix rotMat;
          rotMat.performMatrixMultiplication<false,false>(detectorMatrix,rotationMatrixK);
          doMatVec<false>(rotMat, _srcPts, _dstPts[i]);
          _dstPts[i].x = _dstPts[i].x + center[0];
          _dstPts[i].y = _dstPts[i].y + center[1];
          _dstPts[i].z = 0;
        }

        const double destPoints[3][2]{{_dstPts[0].x, _dstPts[0].y},
                                      {_dstPts[1].x, _dstPts[1].y},
                                      {_dstPts[2].x, _dstPts[2].y}};
        double coeffs[2][3];
        computeWarpAffineMatrix(srcPoints, destPoints, coeffs);
        Real _factor = idata.rotMask ? 0 : NAN;
        stat = cublasScale(handle, numVoxel2D, &_factor, d_projectionAverage, 1);
        NppStatus status = warpAffine(d_projection,
                                      sizeImage,
                                      voxel[1] * sizeof(Real),
                                      rect,
                                      d_projectionAverage,
                                      voxel[1] * sizeof(Real),
                                      rect,
                                      coeffs,
                                      NPPI_INTER_LINEAR);

        if (status < 0) {
          std::cout << "Image rotation failed with error = " << status << "\n";
          exit(EXIT_FAILURE);
        }
        if (status != NPP_SUCCESS) {
          std::cout << YLW << "[WARNING] Image rotation warning = " << status << NRM << "\n";
        }
#ifdef PROFILING
        {
          END_TIMER(TIMERS::IMAGE_ROTATION)
          START_TIMER(TIMERS::MEMCOPY_GPU_CPU)
        }
#endif


        hostDeviceExchange(&projectionGPUAveraged[(j * idata.kVectors.size()) * numVoxel2D + kstart * numVoxel2D],
                           d_projectionAverage, numVoxel2D,
                           hipMemcpyDeviceToHost);
#ifdef PROFILING
        {
          END_TIMER(TIMERS::MEMCOPY_GPU_CPU)
        }
#endif
      }
#ifdef PROFILING
      {
      END_TIMER(TIMERS::ENERGY)
      }
#endif
    }

    /** Freeing bunch of memories not required now **/
    freeCudaMemory(d_polarizationX);
    freeCudaMemory(d_polarizationY);
    freeCudaMemory(d_polarizationZ);
    freeCudaMemory(d_materialConstants);
    if (idata.scatterApproach == ScatterApproach::FULL) {
      freeCudaMemory(d_scatter3D);
    }
    freeCudaMemory(d_voxelInput);

#ifndef EOC
    freeCudaMemory(d_projection);
    freeCudaMemory(d_projectionAverage);
    freeCudaMemory(d_rotProjection);
    if (idata.rotMask) {
      freeCudaMemory(d_mask);
    }
#endif
#ifdef DUMP_FILES
    delete[] polarizationX;
    delete[] polarizationY;
    delete[] polarizationZ;

#endif
#if (defined(DUMP_FILES) or defined(EOC))
    delete[] scatter3D;
#endif

    for(int i = 0; i < NUM_STREAMS; i++) {
      hipfftDestroy(plan[i]);
      gpuErrchk(hipStreamDestroy(streams[i]))
    }
    hipblasDestroy(handle);

#ifdef EOC
    delete[] projectionCPU;
#endif
  }


#ifdef PROFILING
  std::cout << "\n\n[INFO] Timings Info\n";
  for(int i = 0; i < TIMERS::MAX; i++){
    std::cout << "[TIMERS] " << std::left << std::setw(20) << timersName[i] << ":" << timings[i] << " s\n";
  }
  std::cout << "\n\n";
#endif


  return (EXIT_SUCCESS);
}

int cudaMainStreams(const UINT *voxel,
                    const InputData &idata,
                    const std::vector<Material > &materialInput,
                    Real *projectionGPUAveraged,
                    RotationMatrix & rotationMatrix,
                    const Voxel *voxelInput){

  if ((static_cast<uint64_t>(voxel[0]) * voxel[1] * voxel[2]) > std::numeric_limits<BigUINT>::max()) {
    std::cout << "Exiting. Compile by Enabling 64 Bit indices\n";
    exit(EXIT_FAILURE);
  }

  const BigUINT numVoxels = voxel[0] * voxel[1] * voxel[2]; /// Voxel size
  const UINT numVoxel2D = voxel[0] * voxel[1];
  const uint3 vx{voxel[0], voxel[1], voxel[2]};
  const UINT
    numAnglesRotation = static_cast<UINT>(std::round((idata.endAngle - idata.startAngle) / idata.incrementAngle + 1));
  const UINT &numEnergyLevel = idata.energies.size();

  const int & NUM_MATERIAL = idata.NUM_MATERIAL;

  int num_gpu;
  hipGetDeviceCount(&num_gpu);
  std::cout << "Number of CUDA devices:" << num_gpu << "\n";

  if (num_gpu < 1) {
    std::cout << "No GPU found. Exiting" << "\n";
    return (EXIT_FAILURE);
  }

#ifdef PROFILING
  enum TIMERS:UINT{
    MALLOC = 0,
    MEMCOPY_CPU_GPU = 1,
    NtComputation = 2,
    POLARIZATION = 3,
    FFT = 4,
    SCATTER3D = 5,
    IMAGE_ROTATION= 6,
    MEMCOPY_GPU_CPU = 7,
    FREE_MEMORY = 8,
    ENERGY=9,
    MAX = 10
  };
  static const char *timersName[]{"Malloc on CPU + GPU",
                                  "Memcopy CPU -> GPU",
                                   "Nt",
                                  "Polarization",
                                  "FFT",
                                  "Scatter3D + Ewalds",
                                  "Rotation",
                                  "Memcopy GPU -> CPU",
                                  "Free memory",
                                  "Total time "};
  static_assert(sizeof(timersName) / sizeof(char*) == TIMERS::MAX,
                "sizes dont match");
  std::array<std::chrono::high_resolution_clock::time_point,TIMERS::MAX> timerArrayStart;
  std::array<std::chrono::high_resolution_clock::time_point,TIMERS::MAX> timerArrayEnd;
  std::array<Real,TIMERS::MAX> timings{};
  timings.fill(0.0);

#endif

#ifdef DUMP_FILES
  createDirectory("Polarize");
  createDirectory("FFT");
  createDirectory("Scatter");
  createDirectory("Ewald");

  /** Writing VTI files as a cross check **/


  const char * varnameVector[4] = {"material1_s","material2_s","material3_s","material4_s"};
  const char * varnameScalar[4] = {"phi0","phi1", "phi2", "phi3"};

  VTI::writeVoxelDataVector(voxelInput, voxel, "S1", varnameVector,NUM_MATERIAL);
  VTI::writeVoxelDataScalar(voxelInput, voxel, "Phi", varnameScalar,NUM_MATERIAL);
#endif

  omp_set_num_threads(num_gpu);
#pragma omp parallel
  {


    hipSetDevice(omp_get_thread_num());
    hipDeviceProp_t dprop;
    hipGetDeviceProperties(&dprop, omp_get_thread_num());

#ifdef PROFILING
    if(warmup() == EXIT_SUCCESS){
      std::cout << "Warmup completed on GPU " << dprop.name << "\n";
    }
    else{
      std::cout << "Warmup failed on GPU " << dprop.name << "\n";
#pragma omp cancel parallel
      exit (EXIT_FAILURE);
    }
#endif
    static constexpr int NUM_FFT_STREAMS = 3;
    const int NUM_STREAMS = std::max(idata.numMaxStreams,NUM_FFT_STREAMS); // We need minimum of 3 streams for FFT
    std::vector<hipStream_t> streams(NUM_STREAMS);
    hipfftResult result[NUM_FFT_STREAMS];
    hipfftHandle plan[NUM_FFT_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
      gpuErrchk(hipStreamCreate(&streams[i]));
    }
    for(int i = 0; i < NUM_FFT_STREAMS; i++){
      hipfftPlan3d(&plan[i], voxel[2], voxel[1], voxel[0], fftType);
      hipfftSetStream(plan[i],streams[i]);
    }
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hipblasCreate(&handle);

    NppiSize sizeImage;
    sizeImage.height = voxel[0];
    sizeImage.width = voxel[1];

    NppiRect rect;
    rect.height = voxel[0];
    rect.width = voxel[1];
    rect.x = 0;
    rect.y = 0;


    const UINT ompThreadID = omp_get_thread_num();
    const UINT numEnergyPerGPU = static_cast<UINT>(std::ceil(numEnergyLevel * 1.0 / num_gpu));
    const UINT numStart = (numEnergyPerGPU * ompThreadID);
    UINT numEnd = (numEnergyPerGPU * (ompThreadID + 1));
    numEnd = std::min(numEnd, numEnergyLevel);

    const Real &energyStart = numStart < numEnergyLevel ? idata.energies[numStart] : 0;
    const Real &energyEnd = idata.energies[numEnd - 1];

    if (numStart >= numEnergyLevel) {
      std::cout << "[INFO] [GPU = " << dprop.name << "] -> No computation. Idle\n";
    } else {
      std::cout << "[INFO] [GPU = " << dprop.name << "] : " << energyStart << "eV -> " << energyEnd << "eV\n";
    }


#ifdef PROFILING
    {
      START_TIMER(TIMERS::MALLOC);
    }
#endif
#ifdef DUMP_FILES
    Complex *polarizationZ = new Complex[numVoxels];
    Complex *polarizationX = new Complex[numVoxels];
    Complex *polarizationY = new Complex[numVoxels];
#endif
#if defined(EOC) or defined(DUMP_FILES)
    Real *scatter3D = new Real[numVoxels];
#endif

#ifdef EOC
    Real *projectionCPU = new Real[BATCH * voxel[0] * voxel[1]];
#else

#endif

    Voxel *d_voxelInput;
    Complex * d_Nt;
    Material * d_materialConstants;

    mallocGPU(d_Nt, numVoxels*6);
    mallocGPU(d_materialConstants, NUM_MATERIAL);
    const UINT perBatchVoxels = ceil(numVoxels/(NUM_STREAMS*1.0));
    std::vector<UINT> batchID(NUM_STREAMS+1);
    batchID[0] = 0;
    for(int i = 1; i < NUM_STREAMS; i++){
      batchID[i] = (i)*perBatchVoxels;
    }
    batchID[NUM_STREAMS] = numVoxels;

#ifdef PROFILING
    {
      END_TIMER(TIMERS::MALLOC)
    }
#endif



    // TODO: Make this async and overlap with computation
    rotationMatrix.initComputation();
    const auto & baseConfigurations = rotationMatrix.getBaseConfigurations();

    const auto & kVectors = idata.kVectors;


    UINT BlockSize  = static_cast<UINT>(ceil(numVoxels * 1.0 / NUM_THREADS));
    UINT BlockSize2 = static_cast<UINT>(ceil(numVoxel2D * 1.0 / NUM_THREADS));

    for (UINT j = numStart; j < numEnd; j++) {
      hostDeviceExchange(d_materialConstants,&materialInput[j*NUM_MATERIAL],NUM_MATERIAL,hipMemcpyHostToDevice);
      const Real &energy = (idata.energies[j]);
      std::cout << " [STAT] Energy = " << energy << " starting " << "\n";
#ifdef PROFILING
      {
        START_TIMER(TIMERS::ENERGY)
        START_TIMER(TIMERS::MALLOC)
      }
#endif
      cudaZeroEntries(d_Nt,numVoxels*6);
      mallocGPU(d_voxelInput, numVoxels);
#ifdef PROFILING
      {
        END_TIMER(TIMERS::MALLOC)
        START_TIMER(TIMERS::NtComputation)
      }
#endif

      for(int streamID = 0; streamID < NUM_STREAMS; streamID++){
        for(int numMat = 0; numMat < NUM_MATERIAL; numMat++){
          hipMemcpyAsync(&d_voxelInput[batchID[streamID]], &voxelInput[numMat*numVoxels + batchID[streamID]],
                     sizeof(Voxel)*(batchID[streamID+1] -  batchID[streamID]), hipMemcpyHostToDevice,streams[streamID]);
          computeNt(d_materialConstants,d_voxelInput,d_Nt,(MorphologyType)idata.morphologyType,BlockSize,numVoxels,batchID[streamID],batchID[streamID+1],numMat,NUM_STREAMS,streams[streamID],NUM_MATERIAL);
        }
      }
      hipDeviceSynchronize();
      gpuErrchk(hipPeekAtLastError());
#ifdef PROFILING
      {
        END_TIMER(TIMERS::NtComputation)
        START_TIMER(TIMERS::FREE_MEMORY)
      }
#endif


      freeCudaMemory(d_voxelInput);
#ifdef PROFILING
      {
        END_TIMER(TIMERS::FREE_MEMORY)
        START_TIMER(TIMERS::MALLOC)
      }
#endif

      Complex *d_polarizationZ, *d_polarizationX, *d_polarizationY;
      Real *d_scatter3D;
      UINT *d_mask;
      mallocGPU(d_polarizationX, numVoxels);
      mallocGPU(d_polarizationY, numVoxels);
      mallocGPU(d_polarizationZ, numVoxels);

      if (idata.scatterApproach == ScatterApproach::FULL) {
        mallocGPU(d_scatter3D, numVoxels);
      }
#ifndef EOC
      Real *d_projection, *d_rotProjection, *d_projectionAverage;
      mallocGPU(d_projection, numVoxel2D);
      mallocGPU(d_rotProjection, numVoxel2D);
      if (idata.rotMask) {
        mallocGPU(d_mask, numVoxel2D);
      }
      mallocGPU(d_projectionAverage, numVoxel2D);
#endif
#ifdef PROFILING
      {
        END_TIMER(TIMERS::MALLOC)
      }
#endif
      for (UINT kID = 0; kID < kVectors.size(); kID++) {
        const auto & baseConfig = baseConfigurations[kID];
        const Real baseRotAngle = baseConfig.baseRotAngle;
        const Matrix & rotationMatrixK = baseConfig.matrix;
        const Real3 &kVec = idata.kVectors[kID];
        cudaZeroEntries(d_projectionAverage, numVoxel2D);
        if (idata.rotMask) {
          cudaZeroEntries(d_mask, numVoxel2D);
        }


        const Real wavelength = static_cast<Real>(1239.84197 / energy);
        const Real kMagnitude = static_cast<Real>(2 * M_PI / wavelength);
        Real Eangle;
        Matrix ERotationMatrix;

        for (UINT i = 0; i < numAnglesRotation; i++) {
          Eangle = static_cast<Real>((baseRotAngle + idata.startAngle + i * idata.incrementAngle) * M_PI / 180.0);
          computeRotationMatrix(kVec, rotationMatrixK, ERotationMatrix, Eangle);
#ifdef PROFILING
          {
            START_TIMER(TIMERS::POLARIZATION)
          }
#endif
          computePolarization(d_Nt,d_polarizationX,d_polarizationY,d_polarizationZ,BlockSize,(ReferenceFrame)idata.referenceFrame,ERotationMatrix,numVoxels);

#ifdef DUMP_FILES

          CUDA_CHECK_RETURN(hipMemcpy(polarizationX,
                                       d_polarizationX,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          CUDA_CHECK_RETURN(hipMemcpy(polarizationZ,
                                       d_polarizationZ,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          CUDA_CHECK_RETURN(hipMemcpy(polarizationY,
                                       d_polarizationY,
                                       sizeof(Complex) * numVoxels,
                                       hipMemcpyDeviceToHost));
          gpuErrchk(hipPeekAtLastError());
          {
            FILE *pX = fopen("polarizeX.dmp", "wb");
            fwrite(polarizationX, sizeof(Complex), numVoxels, pX);
            fclose(pX);
            FILE *pY = fopen("polarizeY.dmp", "wb");
            fwrite(polarizationY, sizeof(Complex), numVoxels, pY);
            fclose(pY);
            FILE *pZ = fopen("polarizeZ.dmp", "wb");
            fwrite(polarizationZ, sizeof(Complex), numVoxels, pZ);
            fclose(pZ);
            std::string dirname = "Polarize/";
            std::string fname = dirname + "polarizationX" + std::to_string(i);
            VTI::writeDataScalar(polarizationX, voxel, fname.c_str(), "polarizeX");
            fname = dirname + "polarizationY" + std::to_string(i);
            VTI::writeDataScalar(polarizationY, voxel, fname.c_str(), "polarizeY");
            fname = dirname + "polarizationZ" + std::to_string(i);
            VTI::writeDataScalar(polarizationZ, voxel, fname.c_str(), "polarizeZ");
          }
#endif

#ifdef PROFILING
          {
            END_TIMER(TIMERS::POLARIZATION)
            START_TIMER(TIMERS::FFT)
          }
#endif
          /** FFT Computation **/
          result[0] = performFFT(d_polarizationX, plan[0]);
          result[1] = performFFT(d_polarizationY, plan[1]);
          result[2] = performFFT(d_polarizationZ, plan[2]);

          performFFTShift(d_polarizationX, BlockSize, vx,streams[0]);
          performFFTShift(d_polarizationY, BlockSize, vx,streams[1]);
          performFFTShift(d_polarizationZ, BlockSize, vx,streams[2]);
          hipDeviceSynchronize();

          if ((result[0] != HIPFFT_SUCCESS) or (result[1] != HIPFFT_SUCCESS) or (result[2] != HIPFFT_SUCCESS)) {
            std::cout << "CUFFT failed with result " << result[0] << " " << result[1] << " " << result[2] << "\n";
#pragma omp cancel parallel
            exit(EXIT_FAILURE);
          }

#ifdef PROFILING
          {
              END_TIMER(TIMERS::FFT)
              START_TIMER(TIMERS::SCATTER3D)
          }
#endif
          cudaZeroEntries(d_rotProjection, numVoxel2D);
          cudaZeroEntries(d_projection, numVoxel2D);

          if (idata.scatterApproach == ScatterApproach::FULL) {

            performScatter3DComputation(d_polarizationX, d_polarizationY, d_polarizationZ, d_scatter3D,kMagnitude,
                                        numVoxels, vx, idata.physSize, idata.if2DComputation(), BlockSize, kVec);

#ifdef DUMP_FILES
            CUDA_CHECK_RETURN(hipMemcpy(scatter3D, d_scatter3D, sizeof(Real) * numVoxels, hipMemcpyDeviceToHost));
            gpuErrchk(hipPeekAtLastError())
            {
              FILE *scatter = fopen("scatter_3D.dmp", "wb");
              fwrite(scatter3D, sizeof(Real), numVoxels, scatter);
              fclose(scatter);
              std::string dirname = "Scatter/";
              std::string fname = dirname + "scatter" + std::to_string(i);
              VTI::writeDataScalar(scatter3D, voxel, fname.c_str(), "scatter3D");
            }

#endif


#ifdef EOC
            CUDA_CHECK_RETURN(hipMemcpy(scatter3D, d_scatter3D, sizeof(Real) * numVoxels, hipMemcpyDeviceToHost));
            gpuErrchk(hipPeekAtLastError());

#ifdef PROFILING
            {

            }
#endif
            computeEwaldProjectionCPU(projectionCPU, scatter3D, vx, eleField.k.x);
#else
            peformEwaldProjectionGPU(d_projection, d_scatter3D, kMagnitude, vx, idata.physSize,
                                     static_cast<Interpolation::EwaldsInterpolation>(idata.ewaldsInterpolation),
                                     idata.if2DComputation(), BlockSize2, kVec);
#ifdef DUMP_FILES
            hostDeviceExchange(projectionGPUAveraged, d_projection, voxel[0] * voxel[1], hipMemcpyDeviceToHost);
            std::string dirname = "Ewald/";
            std::string fname = dirname + "ewlad" + std::to_string(i);
            VTI::writeDataScalar2DFP(projectionGPUAveraged, voxel, fname.c_str(), "ewald");
            FILE *projection = fopen("projection_scatterFull.dmp", "wb");
            fwrite(projectionGPUAveraged, sizeof(Real), numVoxels, projection);
            fclose(projection);
#endif
          } else {
            peformEwaldProjectionGPU(d_projection, d_polarizationX, d_polarizationY, d_polarizationZ, kMagnitude, vx,
                                     idata.physSize,
                                     static_cast<Interpolation::EwaldsInterpolation>(idata.ewaldsInterpolation),
                                     idata.if2DComputation(), BlockSize2, kVec);
#ifdef DUMP_FILES

            hostDeviceExchange(projectionGPUAveraged, d_projection, voxel[0] * voxel[1], hipMemcpyDeviceToHost);
            std::string dirname = "Ewald/";
            std::string fname = dirname + "ewlad" + std::to_string(i);
            VTI::writeDataScalar2DFP(projectionGPUAveraged, voxel, fname.c_str(), "ewald");
            FILE *projection = fopen("projection_scatterPartial.dmp", "wb");
            fwrite(projectionGPUAveraged, sizeof(Real), numVoxels, projection);
            fclose(projection);
#endif
          }


          Real _factor;
          _factor = NAN;

          stat = cublasScale(handle, numVoxel2D, &_factor, d_rotProjection, 1);


          if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS during scaling failed  with status " << stat << "\n";
            exit(EXIT_FAILURE);
          }


#ifdef PROFILING
          {
            END_TIMER(TIMERS::SCATTER3D)
            START_TIMER(TIMERS::IMAGE_ROTATION)
          }
#endif
          const double alpha = cos(-Eangle);
          const double beta = sin(-Eangle);

          /**https://docs.opencv.org/2.4/modules/imgproc/doc/geometric_transformations.html?highlight=warpaffine**/
          const double coeffs[2][3]{
            alpha, beta, static_cast<Real>(((1 - alpha) * voxel[0] / 2 - beta * voxel[1] / 2.)),
            -beta, alpha, static_cast<Real>(beta * voxel[0] / 2. + (1 - alpha) * voxel[1] / 2.)
          };


          NppStatus status = warpAffine(d_projection,
                                        sizeImage,
                                        voxel[1] * sizeof(Real),
                                        rect,
                                        d_rotProjection,
                                        voxel[1] * sizeof(Real),
                                        rect,
                                        coeffs,
                                        NPPI_INTER_LINEAR);

          if (status < 0) {
            std::cout << "Image rotation failed with error = " << status << "\n";
            exit(-1);
          }
          if (status != NPP_SUCCESS) {
            std::cout << YLW << "[WARNING] Image rotation warning = " << status << NRM << "\n";
          }

          if (idata.rotMask) {
            computeRotationMask<<< BlockSize2, NUM_THREADS >>>(d_rotProjection, d_mask, vx);
            hipDeviceSynchronize();
          }

          const Real factor = static_cast<Real>(1.0);
          stat = cublasAXPY(handle, numVoxel2D, &factor, d_rotProjection, 1, d_projectionAverage, 1);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS during sum failed  with status " << stat << "\n";
            exit(EXIT_FAILURE);
          }

#ifdef PROFILING
          {
            END_TIMER(TIMERS::IMAGE_ROTATION)
          }
#endif
#endif
        }
#ifdef PROFILING
        {
          START_TIMER(TIMERS::IMAGE_ROTATION)
        }
#endif
        if (idata.rotMask) {
          averageRotation<<<BlockSize2, NUM_THREADS>>>(d_projectionAverage, d_mask, vx);
          hipDeviceSynchronize();
          gpuErrchk(hipPeekAtLastError());
        } else {
          /// The averaging out for all angles
          const Real alphaFac = static_cast<Real>(1.0 / numAnglesRotation);
          stat = cublasScale(handle, voxel[0] * voxel[1], &alphaFac, d_projectionAverage, 1);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS during averaging failed  with status " << stat << "\n";
            exit(EXIT_FAILURE);
          }
        }
        //// Rotate Image
        hostDeviceExchange(d_projection, d_projectionAverage, numVoxel2D, hipMemcpyDeviceToDevice);
        const double srcPoints[3][2]{{voxel[0] / 2.,  voxel[1] / 2.},
                                     {voxel[0] * 0.5, voxel[1] * 1.0},
                                     {voxel[0] * 1.0, voxel[1] * 0.5}};
        Real3 _dstPts[3], _srcPts;
        double center[2]{voxel[0] / 2., voxel[1] / 2.};
        for (int i = 0; i < 3; i++) {
          _srcPts.x = srcPoints[i][0] - center[0];
          _srcPts.y = srcPoints[i][1] - center[1];
          _srcPts.z = 0;
          const Matrix & detectorMatrix = rotationMatrix.getDetectorRotationMatrix();
          Matrix rotMat;
          rotMat.performMatrixMultiplication<false,false>(detectorMatrix,rotationMatrixK);
          doMatVec<false>(rotMat, _srcPts, _dstPts[i]);
          _dstPts[i].x = _dstPts[i].x + center[0];
          _dstPts[i].y = _dstPts[i].y + center[1];
          _dstPts[i].z = 0;
        }

        const double destPoints[3][2]{{_dstPts[0].x, _dstPts[0].y},
                                      {_dstPts[1].x, _dstPts[1].y},
                                      {_dstPts[2].x, _dstPts[2].y}};
        double coeffs[2][3];
        computeWarpAffineMatrix(srcPoints, destPoints, coeffs);
        Real _factor = idata.rotMask ? 0 : NAN;
        stat = cublasScale(handle, numVoxel2D, &_factor, d_projectionAverage, 1);
        NppStatus status = warpAffine(d_projection,
                                      sizeImage,
                                      voxel[1] * sizeof(Real),
                                      rect,
                                      d_projectionAverage,
                                      voxel[1] * sizeof(Real),
                                      rect,
                                      coeffs,
                                      NPPI_INTER_LINEAR);

        if (status < 0) {
          std::cout << "Image rotation failed with error = " << status << "\n";
          exit(EXIT_FAILURE);
        }
        if (status != NPP_SUCCESS) {
          std::cout << YLW << "[WARNING] Image rotation warning = " << status << NRM << "\n";
        }
#ifdef PROFILING
        {
          END_TIMER(TIMERS::IMAGE_ROTATION)
          START_TIMER(TIMERS::MEMCOPY_GPU_CPU)
        }
#endif

        hostDeviceExchange(&projectionGPUAveraged[(j * idata.kVectors.size()) * numVoxel2D + kID * numVoxel2D],
                           d_projectionAverage, numVoxel2D,
                           hipMemcpyDeviceToHost);

      }
#ifdef PROFILING
      {
        END_TIMER(TIMERS::MEMCOPY_GPU_CPU)
        START_TIMER(TIMERS::FREE_MEMORY)
      }
#endif
      freeCudaMemory(d_polarizationX);
      freeCudaMemory(d_polarizationY);
      freeCudaMemory(d_polarizationZ);
      if (idata.scatterApproach == ScatterApproach::FULL) {
        freeCudaMemory(d_scatter3D);
      }
#ifndef EOC
      freeCudaMemory(d_projection);
      freeCudaMemory(d_projectionAverage);
      freeCudaMemory(d_rotProjection);
      if (idata.rotMask) {
        freeCudaMemory(d_mask);
      }
#endif
#ifdef PROFILING
      {
        END_TIMER(TIMERS::FREE_MEMORY)
        END_TIMER(TIMERS::ENERGY)
      }
#endif
    }


freeCudaMemory(d_Nt);


#ifdef DUMP_FILES
    delete[] polarizationX;
    delete[] polarizationY;
    delete[] polarizationZ;

#endif
#if (defined(DUMP_FILES) or defined(EOC))
    delete[] scatter3D;
#endif
    for(int i = 0; i < NUM_FFT_STREAMS; i++) {
      hipfftDestroy(plan[i]);
    }
    for(int i = 0; i < NUM_STREAMS; i++) {
      gpuErrchk(hipStreamDestroy(streams[i]))
    }
    hipblasDestroy(handle);
#ifdef EOC
    delete[] projectionCPU;
#endif
  }


#ifdef PROFILING
  std::cout << "\n\n[INFO] Timings Info\n";
  for(int i = 0; i < TIMERS::MAX; i++){
    std::cout << "[TIMERS] " << std::left << std::setw(20) << timersName[i] << ":" << timings[i] << " s\n";
  }
  std::cout << "\n\n";
#endif


  return (EXIT_SUCCESS);

}

int computePolarization(const UINT *voxel, const InputData &idata, const std::vector<Material > &materialInput,
                        Complex *polarizationX,Complex *polarizationY,Complex *polarizationZ,
                        RotationMatrix & rotationMatrix, const Voxel *voxelInput, const Real EAngle, const UINT energyID,
                        const int NUM_MATERIAL){

  if ((static_cast<uint64_t>(voxel[0]) * voxel[1] * voxel[2]) > std::numeric_limits<BigUINT>::max()) {
    std::cout << "Exiting. Compile by Enabling 64 Bit indices\n";
    exit(EXIT_FAILURE);
  }

  if(idata.caseType != DEFAULT){
    std::cout << "Only implemented for Case Type = 0\n";
    return EXIT_FAILURE;
  }
  const BigUINT numVoxels = voxel[0] * voxel[1] * voxel[2]; /// Voxel size
  const uint3 vx{voxel[0], voxel[1], voxel[2]};
  const UINT
    numAnglesRotation = static_cast<UINT>(std::round((idata.endAngle - idata.startAngle) / idata.incrementAngle + 1));
  const UINT &numEnergyLevel = idata.energies.size();


  int num_gpu;
  hipGetDeviceCount(&num_gpu);
  std::cout << "Number of CUDA devices:" << num_gpu << "\n";

  if (num_gpu < 1) {
    std::cout << "No GPU found. Exiting" << "\n";
    return (EXIT_FAILURE);
  }
  Material * d_materialConstants;
  Voxel * d_voxelInput;
  Complex *d_polarizationZ, *d_polarizationX, *d_polarizationY;
  mallocGPU(d_polarizationX, numVoxels);
  mallocGPU(d_polarizationY, numVoxels);
  mallocGPU(d_polarizationZ, numVoxels);
  mallocGPU(d_voxelInput,numVoxels*NUM_MATERIAL);
  mallocGPU(d_materialConstants,NUM_MATERIAL);

  UINT BlockSize  = static_cast<UINT>(ceil(numVoxels * 1.0 / NUM_THREADS));

  hostDeviceExchange(d_voxelInput, voxelInput, numVoxels*NUM_MATERIAL, hipMemcpyHostToDevice);
  hostDeviceExchange(d_materialConstants, &materialInput[energyID*NUM_MATERIAL],NUM_MATERIAL, hipMemcpyHostToDevice);

  // TODO: Make this async and overlap with computation
  rotationMatrix.initComputation();
  const auto & baseConfigurations = rotationMatrix.getBaseConfigurations();

  const int kID = 0;
  const auto & baseConfig = baseConfigurations[kID];
  const Matrix & rotationMatrixK = baseConfig.matrix;
  const Real3 &kVec = idata.kVectors[kID];
  Matrix ERotationMatrix;
  computeRotationMatrix(kVec, rotationMatrixK, ERotationMatrix, EAngle);
  computePolarization(d_materialConstants, d_voxelInput, vx, d_polarizationX, d_polarizationY,
                      d_polarizationZ, static_cast<FFT::FFTWindowing >(idata.windowingType),
                      idata.if2DComputation(), static_cast<MorphologyType>(idata.morphologyType), BlockSize,
                      static_cast<ReferenceFrame>(idata.referenceFrame), ERotationMatrix,numVoxels,idata.NUM_MATERIAL);

  hostDeviceExchange(polarizationX,d_polarizationX,numVoxels,hipMemcpyDeviceToHost);
  hostDeviceExchange(polarizationY,d_polarizationY,numVoxels,hipMemcpyDeviceToHost);
  hostDeviceExchange(polarizationZ,d_polarizationZ,numVoxels,hipMemcpyDeviceToHost);

  freeCudaMemory(d_voxelInput);
  freeCudaMemory(d_polarizationX);
  freeCudaMemory(d_polarizationY);
  freeCudaMemory(d_polarizationZ);
  freeCudaMemory(d_materialConstants);

  return EXIT_SUCCESS;
}